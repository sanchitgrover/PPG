#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <stdint.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cycleTimer.h"
#include "utils.h"

#define DEBUG           1 
#if DEBUG
#define cudaCheckError(ans)  cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

#define METHOD          2
#define USE_FIRST_PR    1
#define SKIP_EVEN       1

#define THREADS_PER_BLK 128
#define SHARED_MEM_SIZE 8192
#define NUM_STREAMS     16
#define NUM_FIRST_PR    512 


//void print_primes(bool *masks, uint64_t lo, 
//                uint64_t n, bool valForPrime)

#if METHOD == 1
__global__ void soe(bool *masks, uint64_t n, uint64_t k)
{
    // Ignore idx = 0 and 1 so add 2
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + 2;

    // Mark 0 and 1 as non primes
    if (idx == 2) {
        masks[0] = true;
        masks[1] = true;
    }

    if (idx <= k) {
        // If current number is unmarked
        if (!masks[idx]) {
            // Mark all multiples of i
            // Start from 2i since we don't want to mark i
            for(int j=2*idx; j<=n; j+=idx) {
                masks[j] = true;
            } 
        }
    }
}
#elif METHOD == 2
#if USE_FIRST_PR
__global__ void soe(bool *masks, bool *primeMasks, uint64_t lo,
        uint64_t chunkLo, uint64_t n, uint64_t k)
#else
__global__ void soe(bool *masks, uint64_t lo, uint64_t chunkLo,
        uint64_t n, uint64_t k)
#endif
{
    __shared__ bool cache[SHARED_MEM_SIZE];

#if USE_FIRST_PR
    __shared__ bool firstPrimes[NUM_FIRST_PR+1];
#endif

    uint64_t tid = threadIdx.x;
    uint64_t off = blockIdx.x * blockDim.x;

    uint64_t blockPos = (blockIdx.x * SHARED_MEM_SIZE);
    uint64_t startPos = chunkLo + blockPos;
    uint64_t endPos = startPos + SHARED_MEM_SIZE;
    endPos = (n < endPos) ? n+1 : endPos;

    //printf("%lu %" PRIu64 " %" PRIu64 "\n", blockIdx.x, startPos, endPos); 
    uint64_t i, j;

    // Load respective portion from array
    for (i=startPos+tid; i<endPos; i+=THREADS_PER_BLK) {
        cache[i-startPos] = masks[i - chunkLo];
#if USE_FIRST_PR
        // Init firstPrimes array upto SHARED_MEM_SIZE
        //firstPrimes[i-startPos] = false;
#endif
        if (i - chunkLo > CHUNK_SIZE) {
            printf("LD cl:%"PRIu64",sp:%"PRIu64"\n",chunkLo, startPos);
        }
    }
#if USE_FIRST_PR
    // Initialize rest of the indices for firstPrimes
    for (i=tid; i<=NUM_FIRST_PR; i+=THREADS_PER_BLK) {
        firstPrimes[i] = primeMasks[i];
    }
#endif
    __syncthreads();

    // Only first thread should mark 0 and 1 as non primes
    // This happens when startPos = 0 and off+tid = 0
    if (!(startPos | (off + tid))) {
        cache[0] = true;
#if SKIP_EVEN == 0
        cache[1] = true;
#endif
    }

#if SKIP_EVEN
    i = (2*tid) + 3;
    for ( ; i<=k; i+= 2*THREADS_PER_BLK) {
#else
    for (i=tid+2; i<=k; i+= THREADS_PER_BLK) {
#endif
#if USE_FIRST_PR
        // If current number is unmarked
        // Start from 2 since 0 and 1 are not primes
        // Chedk only till the size of the array
#if SKIP_EVEN
        if ((i > NUM_FIRST_PR) || (!firstPrimes[i/2])) {
#else
        if ((i > NUM_FIRST_PR) || (!firstPrimes[i])) {
#endif
#endif
#if SKIP_EVEN
            // Find first multiple of i within the range
            uint64_t firstMult = (((2*startPos) + 1) / i) * i;
            // Need to get first odd since indices for even don't exist
            firstMult = (firstMult % 2) ? firstMult : firstMult + i;
            // and convert it back to correct range
            firstMult /= 2;//(firstMult) ? ((firstMult-1)/2) : (i/2);
#else
            // Find first multiple of i within the range
            uint64_t firstMult = (startPos / i) * i;
#endif
            // Mark all multiples of i within the range
            for(j=firstMult; j<endPos; j+=i) {
                // Do not mark i itself
#if SKIP_EVEN
                // Compare converted value of i with j
                if ((i/2 != j) && (j >= startPos)) {
#else
                if ((i != j) && (j >= startPos)) {
#endif
                    cache[j-startPos] = true;
                    //printf("%lu, %lu, s: %lu, e:%lu\n", i, j, startPos, endPos);
#if 0
                    if (j <= NUM_FIRST_PR) {
                        firstPrimes[j-2] = true;
                    }
#endif
                    if (j-startPos > SHARED_MEM_SIZE) {
                        printf("%"PRIu64"\n",j-startPos);
                    }
                }
            }   
#if USE_FIRST_PR
        }   
#endif
    }

    __syncthreads();

    // Copy back to global memory
    for (i=startPos + tid; i<endPos; i+=THREADS_PER_BLK) {
        masks[i - chunkLo] = cache[i-startPos];
        if (i - chunkLo > CHUNK_SIZE) {
            printf("ST cl:%"PRIu64",sp:%"PRIu64"\n",chunkLo, startPos);
            //printf("ST %"PRIu64"\n",startPos - chunkLo + i);
        }
    }
}
#endif

__global__ void print_arr(bool *arr, uint64_t lo, uint64_t chunkLo, uint64_t hi
        , bool cmp) 
{
#if SKIP_EVEN
    if((chunkLo <= 2) && (hi >= 2)) {
        // Print 2
        printf("2\n");
    }
#endif
#if 0
    // Find first odd number close to chunkLo
    uint64_t j = (chunkLo % 2) ? chunkLo : chunkLo + 1;
    for (; j<=hi; j+=2) {
#else
    for (uint64_t j=chunkLo; j<=hi; j++) {
#endif
        if(arr[j-chunkLo] == cmp) {
#if SKIP_EVEN
            printf("%"PRIu64"\n", (2*j) + 1); 
#else
            printf("%"PRIu64"\n", j);
#endif
        }   
    }   
}

/*void init_dev_masks(bool *devMasks, int n)
{
    // We should be able to mark elements upto n
    // Therefore, index can go upto n, requiring n+1 elements
    size_t maskSize = sizeof(bool) * (n + 1); 
    hipMalloc(&devMasks, maskSize);
    hipMemset(devMasks, 0, maskSize);
}

void deinit_dev_masks(bool *devMasks)
{
    hipFree(devMasks);
}*/
#if USE_FIRST_PR
__global__ void soa(bool *masks, bool *primeMasks, uint64_t lo, uint64_t chunkLo,
        uint64_t n, uint64_t k)
#else
__global__ void soa(bool *masks, uint64_t lo, uint64_t chunkLo, uint64_t n,
        uint64_t k)
#endif
{
    __shared__ int cache[SHARED_MEM_SIZE];
    
    // LUTs for calculating modulo values
    __shared__ bool arr1[MOD_VAL];
    __shared__ bool arr2[MOD_VAL];
    __shared__ bool arr3[MOD_VAL];
    
#if USE_FIRST_PR
    __shared__ bool firstPrimes[NUM_FIRST_PR+1];
#endif

    uint64_t tid = threadIdx.x;
    uint64_t off = blockIdx.x * blockDim.x;

    uint64_t blockPos = (blockIdx.x * SHARED_MEM_SIZE);
    uint64_t startPos = chunkLo + blockPos;
    uint64_t endPos = startPos + SHARED_MEM_SIZE;
    endPos = (n < endPos) ? n+1 : endPos;

    //printf("%lu %" PRIu64 " %" PRIu64 "\n", blockIdx.x, startPos, endPos); 
    uint64_t i;

    // Load respective portion from array
    for (i=startPos+tid; i<endPos; i+=THREADS_PER_BLK) {
        cache[i-startPos] = (masks[i - chunkLo]) ? 1 : 0;
        if (i - chunkLo > CHUNK_SIZE) {
            printf("LD cl:%"PRIu64",sp:%"PRIu64"\n",chunkLo, startPos);
        }   
    }   

#if USE_FIRST_PR
    // Initialize rest of the indices for firstPrimes
    for (i=tid; i<=NUM_FIRST_PR; i+=THREADS_PER_BLK) {
        firstPrimes[i] = primeMasks[i];
    }   
#endif

    for (i=tid; i<MOD_VAL; i+=THREADS_PER_BLK) {
        arr1[i] = false;
        arr2[i] = false;
        arr3[i] = false;
    }

    __syncthreads();

    if (tid == 0) {
        // Set appropriate indices to true
        arr1[1] = true;
        arr1[13] = true;
        arr1[17] = true;
        arr1[29] = true;
        arr1[37] = true;
        arr1[41] = true;
        arr1[49] = true;
        arr1[53] = true;

        arr2[7] = true;
        arr2[19] = true;
        arr2[31] = true;
        arr2[43] = true;

        arr3[11] = true;
        arr3[23] = true;
        arr3[47] = true;
        arr3[59] = true;
    }
    
    if (!((tid + off) | startPos)) {  
        // Mark 2, 3 and 5 as primes
        cache[2] = 1;
        cache[3] = 1;
        cache[5] = 1;
    }
    __syncthreads();

    // For solutions to 4x^2 + y^2 = n within the range,
    // max x and y are as follows
    uint64_t xmax = ceilf(sqrtf(endPos)/2);
    uint64_t ymax = ceilf(sqrtf(endPos));
    uint64_t x = 1;
    uint64_t y = 1;
    uint64_t x2 = 1;
    uint64_t y2 = 1;
    uint64_t pdt;
    uint8_t r;

    // Solutions for 4x^2 + y^2 = n
    for (x=tid+1; x*x<endPos; x+=THREADS_PER_BLK) {
        x2 = x * x;
        for (y=1; y*y<endPos; y+=2) {
            y2 = y * y;
            pdt = (4*x2) + y2;
            r = pdt % 60;
            if (pdt >= endPos) {
                // Break if this happens as y is increasing so 
                // pdt will keep increasing for same x
                break;
            }
            if (pdt >= startPos) {
                if(arr1[r]) {
                    // Flip bit
                   //cache[pdt-startPos] ^= true;
                    atomicXor(&cache[pdt-startPos], 1);
                }
            }
        }
    }
    
    // Solutions for 3x^2 + y^2
    xmax = ceilf(sqrtf(endPos/3));
   
    // Odd xs even ys
    for (x=((2*tid)+1); x*x<endPos; x+=THREADS_PER_BLK) {
        x2 = x * x;
        for (y=2; y*y<endPos; y+=2) {
            y2 = y * y;
            pdt = (3*x2) + y2; 
            r = pdt % 60; 
            if (pdt >= endPos) {
                // Break if this happens as y is increasing so 
                // pdt will keep increasing for same x
                break;
            }   
            if (pdt >= startPos) {
                if(arr2[r]) {
                    // Flip bit
                    atomicXor(&cache[pdt-startPos], 1);
                }   
            }   
        }   
    }
    
    // Solutions for 3x^2 - y^2
    // Max x will be when y = x because x > y condition has to be met
    xmax = ceilf(sqrtf(endPos/2));
    
    for (x=tid+1; x*x<endPos; x+=THREADS_PER_BLK) {
        x2 = x * x;
        for (i=1; i<x; i+=2) {
            y = x - i; // Get odd/even combos
            y2 = y * y;
            pdt = (3*x2) - y2;
            r = pdt % 60;
            if (pdt >= endPos) {
                // Break if this happens as y is decreasing so 
                // pdt will keep increasing for same x
                break;
            }
            if (pdt >= startPos) {
                if(arr3[r]) {
                    // Flip bit
                    atomicXor(&cache[pdt-startPos], 1);
                    //cache[pdt-startPos] ^= true;
                }
            }
        }
    }
    
    __syncthreads();

    // Mark multiples of squares
    uint64_t i2 = 4;
    uint64_t imax = ceilf(sqrtf(endPos)); 

    for (i=tid+2; i*i<endPos; i+=THREADS_PER_BLK) {
        i2 = i * i;
        
#if USE_FIRST_PR
        if ((i > NUM_FIRST_PR) || (!firstPrimes[i])) {
#endif
            // Find first multiple within range
            uint64_t firstMult = (startPos / i2) * i2;

            // If prime, mark all multiples of square of that prime
            for (uint64_t j=firstMult; j < endPos; j += i2) {
                if (j >= startPos) {
                    cache[j-startPos] = 0;
                }
            }
#if USE_FIRST_PR
        }
#endif
    }

    __syncthreads();

    // Copy back to global memory
    for (i=startPos + tid; i<endPos; i+=THREADS_PER_BLK) {
        masks[i - chunkLo] = (cache[i-startPos]) ? true : false;
        if (i - chunkLo > CHUNK_SIZE) {
            printf("ST cl:%"PRIu64",sp:%"PRIu64"\n",chunkLo, startPos);
            //printf("ST %"PRIu64"\n",startPos - chunkLo + i);
        }
    }

}

void par_algo(bool *devMasks, bool *primeMasks, uint64_t lo, uint64_t chunkLo,
        uint64_t n, uint64_t k, hipStream_t stream, bool isSOA)
{
    //uint64_t k = floor(sqrt(n));
#if METHOD == 1
    int numBlks = (k / THREADS_PER_BLK) + 1;
#elif METHOD == 2
    int numBlks = (CHUNK_SIZE / SHARED_MEM_SIZE) + 1;
    if (n - chunkLo < CHUNK_SIZE) {
        numBlks = ((n - chunkLo) / SHARED_MEM_SIZE) + 1;
    }
#endif
    if (isSOA) {
#if USE_FIRST_PR
        soa<<<numBlks, THREADS_PER_BLK, 0, stream>>>(devMasks, primeMasks, lo,
                chunkLo, n, k);
#else
        soa<<<numBlks, THREADS_PER_BLK, 0, stream>>>(devMasks, lo, chunkLo, n, k);
#endif
    } else {
#if METHOD == 1
    soe<<<numBlks, THREADS_PER_BLK, 0, stream>>>(devMasks, n, k);
#else
#if USE_FIRST_PR
        soe<<<numBlks, THREADS_PER_BLK, 0, stream>>>(devMasks, primeMasks, lo,
                chunkLo, n, k);
#else
        soe<<<numBlks, THREADS_PER_BLK, 0, stream>>>(devMasks, lo, chunkLo, n, k);
#endif
#endif
    }
}

void par_find_primes(bool *hostMasks, uint64_t lo, uint64_t n, uint64_t k, bool isSOA,
        bool isDisplayTime, bool isQuiet, bool primeBoolVal)
{
    bool *devMasks[NUM_STREAMS];
    hipStream_t streams[NUM_STREAMS]; 
    size_t maskSize = sizeof(bool) * (CHUNK_SIZE + 1); 
  
    bool *primeMasks;

    // Effective n (for skipping evens will be different)
#if SKIP_EVEN
    uint64_t nEff = (n-1)/2;
#else
    uint64_t nEff = n;
#endif

#if USE_FIRST_PR
    hipMalloc(&primeMasks, sizeof(bool) * (NUM_FIRST_PR+1));
    hipMemset(primeMasks, 0, sizeof(bool) * (NUM_FIRST_PR+1));
#endif

    // We should be able to mark elements upto n
    // Therefore, index can go upto n, requiring n+1 elements
    for (int i=0; i<NUM_STREAMS; i++) {
        hipMalloc(&devMasks[i], maskSize);
        hipMemset(devMasks[i], 0, maskSize);
        cudaCheckError(hipStreamCreate(&streams[i]));
    }
    // Performance metrics for parallel execution
    hipEvent_t start, stop;//, stop_mem;
    float duration;//, duration_mem;

    if (isDisplayTime) {
        // Init objects
        hipEventCreate(&start);
        hipEventCreate(&stop);
        //hipEventCreate(&stop_mem);

        // Start recording time
        hipEventRecord(start);
    }

#if METHOD == 2
#if USE_FIRST_PR
    soe<<<(NUM_FIRST_PR/CHUNK_SIZE) + 1, THREADS_PER_BLK>>>(primeMasks, 
            primeMasks, 0, 0, NUM_FIRST_PR, ceilf(sqrtf(NUM_FIRST_PR)));
    //print_arr<<<1,1>>>(primeMasks, 0, 0, NUM_FIRST_PR, false);
    //hipDeviceSynchronize();
#endif
#endif
    int ctr = 0;
    for (uint64_t i=lo; i<=nEff; i+=CHUNK_SIZE) {
        uint64_t endPos = i + CHUNK_SIZE;
        endPos = (nEff < endPos) ? nEff : endPos;
        
        int streamId = ctr % NUM_STREAMS;
        hipStream_t stream = streams[streamId];
        //printf("Stream %d\n", streamId);
        par_algo(devMasks[streamId], primeMasks, lo, i, endPos, k, stream, isSOA);
        
        if (!isQuiet) {
            print_arr<<<1, 1, 0, stream>>>(devMasks[streamId], lo, i, endPos, primeBoolVal);
        }
        ctr++;
        if (ctr >= NUM_STREAMS) {
            cudaCheckError(hipStreamSynchronize(streams[ctr % NUM_STREAMS]));
            cudaCheckError(hipMemsetAsync(devMasks[ctr % NUM_STREAMS], 0,
                        maskSize, streams[ctr % NUM_STREAMS]));
        }
    }
    
    if (isDisplayTime) {
        hipEventRecord(stop);
    }

    //hipMemcpy(hostMasks, devMasks, maskSize, hipMemcpyDeviceToHost);
    
    if (isDisplayTime) {
        //hipEventRecord(stop_mem);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&duration, start, stop);
        printf("Time taken by GPU for generating primes upto %lu: %fms\n",
                n, duration);

        //hipEventSynchronize(stop_mem);
        //hipEventElapsedTime(&duration_mem, start, stop_mem);
        //printf("Time taken by GPU for generating primes upto %lu, including"
        //        " memcopy: %fms\n", n, duration_mem);
    }

    for (int i=0; i<NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
        hipFree(devMasks[i]);
    }
}
